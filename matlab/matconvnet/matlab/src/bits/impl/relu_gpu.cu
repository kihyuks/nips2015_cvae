#include "hip/hip_runtime.h"
// @file relu_gpu.cu
// @brief Relu non-linearity implementation (GPU)
// @author Xinchen Yan
// Created by Xinchen Yan, Apr. 23, 2015 (in compatible with matconvnet 1.0.9)

#include "relu.hpp"
#include "../datacu.hpp"
#include <assert.h>
#include <float.h>
#include <sm_20_atomic_functions.h>

/* ------------------------------------------------------------ */
/*                                                   Relu (GPU) */
/* ------------------------------------------------------------ */

template<typename T> __global__ void
relu_gpu_kernel(
  T* output,
  const T* data,
  const int nthreads,
  const int width,
  const int height)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  T zero = (T)0;
  if (index < nthreads) {
    //int x = index;
    //int y = x / width;
    //int z = y / height;
    //x %= width;
    //y %= height;

    //output += z * (width*height);

    if (data[index] > zero)
      output[index] = data[index];
    else
      output[index] = zero;
  }

}

template<> vl::Error
vl::impl::relu_forward<vl::GPU, float>(float* output, float const* data, 
                                       size_t height, size_t width, size_t depth)
{
  int nthreads = width * height * depth;

  relu_gpu_kernel<float>
  <<< divideUpwards(nthreads, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (output, data,
   nthreads, height, width);

  hipError_t status = hipPeekAtLastError();
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}

/* ------------------------------------------------------------ */
/*                                           ReluBackward (GPU) */
/* ------------------------------------------------------------ */

template <typename T> __global__ void
relu_backward_gpu_kernel(
  T* derData,
  const T* data,
  const T* derOutput,
  const int nthreads,
  const int width,
  const int height)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  T zero = (T)0;
  if (index < nthreads) {
    
    //int x = index;
    //int y = x / width;
    //int z = y / height;
    //x %= width;
    //y %= height;

    if (data[index] > zero)
      derData[index] = derOutput[index];
    else
      derData[index] = zero;
  }
}

template<> vl::Error
vl::impl::relu_backward<vl::GPU, float>(float* derData, float const* data, float const* derOutput,
                                        size_t height, size_t width, size_t depth)
{
  int nthreads = width * height * depth;
  
  relu_backward_gpu_kernel<float>
  <<< divideUpwards(nthreads, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (derData, data, derOutput,
   nthreads, height, width);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}
