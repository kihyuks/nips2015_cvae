// @file nnunpooling_cudnn.cu
// @brief Unpooling block CuDNN-based implementation.
// @author Xinchen
// Modified by Xinchen Yan, Feb. 19, 2015

#if !defined(ENABLE_GPU) | !defined(ENABLE_CUDNN)
#error "nnunpooling_cudnn.hpp cannot be compiled without GPU and CUDNN support."
#endif

#include "nnunpooling_cudnn.hpp"
#include "../datacu.hpp"
#include <assert.h>

using namespace vl ;

#define CHECK(x) \
{ \
cudnnError = x ; \
if (cudnnError != HIPDNN_STATUS_SUCCESS) { \
error = context.setError(context.getCudaHelper().catchCudnnError(cudnnError, \
STRINGIZE(__LINE__) ":" STRINGIZE(__FILE__))) ; \
goto done ; \
} }

/* nnunpooling_forward_cudnn */
template<> vl::Error
vl::impl::nnunpooling_forward_cudnn<float>(Context& context,
                                           Tensor output,
                                           Tensor data,
                                           int strideY, int strideX)

{
  assert(output) ;
  assert(data) ;
  
  hipdnnTensorDescriptor_t outputDesc, dataDesc ;
  cudnn
//TODO

}



