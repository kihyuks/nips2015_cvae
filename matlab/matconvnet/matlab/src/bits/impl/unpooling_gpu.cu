#include "hip/hip_runtime.h"
// @file unpooling_gpu.cu
// @brief Unpooling block implementation (GPU)
// @author Xinchen Yan
// Modified by Xinchen, Feb. 19, 2015 (in compatible with matconvnet 1.0.9)

#include "unpooling.hpp"
#include "../datacu.hpp"
#include <assert.h>
#include <float.h>
#include <sm_20_atomic_functions.h>

/* ---------------------------------------------------------------- */
/*                                                Unpooling (GPU) */
/* ---------------------------------------------------------------- */
using namespace std;
template<typename T> __global__ void 
unpooling_gpu_kernel
(T* unpooled,
 const T* data,
 const int nthreads,
 const int width,
 const int height,
 const int strideX,
 const int strideY)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    int unpooledWidth = width * strideX;
    int unpooledHeight = height * strideY;
    int x = index ;
    int y = x / width ;
    int z = y / height ;
    x %= width ;
    y %= height ;

    unpooled += z * (unpooledWidth*unpooledHeight) ;

    int x1 = x * strideX;
    int y1 = y * strideY;
    int x2 = (x + 1) * strideX;
    int y2 = (y + 1) * strideY;
    for (int v = y1 ; v < y2 ; ++v) {
      for (int u = x1 ; u < x2 ; ++u) {
        unpooled[v * unpooledWidth + u] = (T)0;
      }
    }
    unpooled[y1 * unpooledWidth + x1] = data[index] ;
  }
}


template<> vl::Error
vl::impl::unpooling_forward<vl::GPU, float>(float* unpooled,
                                     float const* data,
                                     size_t height, size_t width, size_t depth,
                                     size_t strideY, size_t strideX)
{
  int nthreads = width * height * depth;

  unpooling_gpu_kernel<float>
  <<< divideUpwards(nthreads, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (unpooled, data,
   nthreads, height, width,
   strideY, strideX);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}

/* ---------------------------------------------------------------- */
/*                                         UnpoolingBackward (GPU) */
/* ---------------------------------------------------------------- */

//#ifdef VLNN_CAFFELIKE_BPPOLL
template <typename T> __global__ void 
unpooling_backward_gpu_kernel(
    T* derData,
    const T* derUnpooled,
    const int nthreads,
    const int width,
    const int height,
    const int strideX,
    const int strideY)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    
    int unpooledWidth = width * strideX;
    int unpooledHeight = height * strideY;

    int x_data = index ;
    int y_data = x_data / width ;
    int z = y_data / height ;
    x_data %= width ;
    y_data %= height ;
    
    derUnpooled += z * (unpooledWidth * unpooledHeight);
    int x1 = x_data * strideX;
    int y1 = y_data * strideY;
    
    derData[index] += derUnpooled[y1 * unpooledWidth + x1];
  }
}
//#endif

template<> vl::Error
vl::impl::unpooling_backward<vl::GPU, float>(float* derData,
                                          float const* derUnpooled,
                                          size_t height, size_t width, size_t depth,
                                          size_t strideY, size_t strideX)
{
  int nthreads = width * height * depth;
  
  unpooling_backward_gpu_kernel<float>
  <<< divideUpwards(nthreads, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (derData, derUnpooled,
   nthreads,
   height, width,
   strideY, strideX);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}

