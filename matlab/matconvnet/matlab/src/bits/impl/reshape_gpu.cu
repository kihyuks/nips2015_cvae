#include "hip/hip_runtime.h"
// @file reshape_gpu.cu
// @brief Reshape implementation (GPU)
// @author Xinchen Yan
// Created by Xinchen Yan, Apr. 23, 2015 (in compatible with matconvnet 1.0.9)

#include "reshape.hpp"
#include "../datacu.hpp"
#include <assert.h>
#include <float.h>
#include <sm_20_atomic_functions.h>

/* ------------------------------------------------------------ */
/*                                                Reshape (GPU) */
/* ------------------------------------------------------------ */

template<typename T> __global__ void
reshape_gpu_kernel(
  T* output,
  const T* data,
  const int nthreads
)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    output[index] = data[index];//TODO: check gradient
  }
}

template<> vl::Error
vl::impl::reshape_forward<vl::GPU, float>(float* output, float const* data, 
                                       size_t ndim, size_t ndata)
{
  int nthreads = ndim * ndata;

  reshape_gpu_kernel<float>
  <<< divideUpwards(nthreads, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (output, data, nthreads);

  hipError_t status = hipPeekAtLastError();
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}

/* ------------------------------------------------------------ */
/*                                        ReshapeBackward (GPU) */
/* ------------------------------------------------------------ */

template <typename T> __global__ void
reshape_backward_gpu_kernel(
  T* derData,
  const T* derOutput,
  const int nthreads)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    derData[index] = derOutput[index];
  }
}

template<> vl::Error
vl::impl::reshape_backward<vl::GPU, float>(float* derData, float const* derOutput,
                                        size_t ndim, size_t ndata)
{
  int nthreads = ndim * ndata;
  
  reshape_backward_gpu_kernel<float>
  <<< divideUpwards(nthreads, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (derData, derOutput, nthreads);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}
