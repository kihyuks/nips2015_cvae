#include "hip/hip_runtime.h"
// @file upsampling_gpu.cu
// @brief Upsampling block implementation (GPU)
// @author Xinchen Yan
// Created by Xinchen, Apr. 23, 2015 (in compatible with matconvnet 1.0.9)
//#include <iostream>
#include "upsampling.hpp"
#include "../datacu.hpp"
#include <assert.h>
#include <float.h>
#include <sm_20_atomic_functions.h>

/* ---------------------------------------------------------------- */
/*                                                 Upsampling (GPU) */
/* ---------------------------------------------------------------- */

//using namespace std;

template<typename T> __global__ void 
upsampling_gpu_kernel
(T* upsampled,
 const T* data,
 const bool sparse,
 const int nthreads,
 const int width,
 const int height,
 const int strideX,
 const int strideY)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    int upsampledWidth = width * strideX;
    int upsampledHeight = height * strideY;
    int x = index ;
    int y = x / width ;
    int z = y / height ;
    x %= width ;
    y %= height ;

    upsampled += z * (upsampledWidth*upsampledHeight) ;

    int x1 = x * strideX;
    int y1 = y * strideY;
    int x2 = (x + 1) * strideX;
    int y2 = (y + 1) * strideY;
    for (int v = y1 ; v < y2 ; ++v) {
      for (int u = x1 ; u < x2 ; ++u) {
        if (sparse)
          upsampled[v * upsampledWidth + u] = (T)0;
        else
          upsampled[v * upsampledWidth + u] = data[index];
      }
    }
    upsampled[y1 * upsampledWidth + x1] = data[index] ;
  }
}


template<> vl::Error
vl::impl::upsampling_forward<vl::GPU, float>(float* upsampled,
                                      float const* data,
                                      bool sparse,
                                      size_t height, size_t width, size_t depth,
                                      size_t strideY, size_t strideX)
{
  int nthreads = width * height * depth;

  //std::cerr << sparse << std::endl;
  upsampling_gpu_kernel<float>
  <<< divideUpwards(nthreads, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (upsampled, data,
   sparse,
   nthreads, height, width,
   strideY, strideX);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}

/* ---------------------------------------------------------------- */
/*                                         UpsamplingBackward (GPU) */
/* ---------------------------------------------------------------- */

//#ifdef VLNN_CAFFELIKE_BPPOLL
template <typename T> __global__ void 
upsampling_backward_gpu_kernel(
    T* derData,
    const T* derUpsampled,
    const bool sparse,
    const int nthreads,
    const int width,
    const int height,
    const int strideX,
    const int strideY)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    
    int upsampledWidth = width * strideX;
    int upsampledHeight = height * strideY;

    int x_data = index ;
    int y_data = x_data / width ;
    int z = y_data / height ;
    x_data %= width ;
    y_data %= height ;
    
    derUpsampled += z * (upsampledWidth * upsampledHeight);
    int x1 = x_data * strideX;
    int y1 = y_data * strideY;
    int x2 = (x_data + 1) * strideX;
    int y2 = (y_data + 1) * strideY;
    if (sparse) {
      derData[index] += derUpsampled[y1 * upsampledWidth + x1];
    } else {
      for (int v = y1 ; v < y2 ; ++v) {
        for (int u = x1 ; u < x2 ; ++u) { 
          derData[index] += derUpsampled[v * upsampledWidth + u];
        }
      }
      derData[index] /= (T)(strideX * strideY);    
    } 
  }
}
//#endif

template<> vl::Error
vl::impl::upsampling_backward<vl::GPU, float>(float* derData,
                                              float const* derUpsampled,
                                              bool sparse,
                                              size_t height, size_t width, size_t depth,
                                              size_t strideY, size_t strideX)
{
  int nthreads = width * height * depth;
  
  upsampling_backward_gpu_kernel<float>
  <<< divideUpwards(nthreads, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
  (derData, derUpsampled,
   sparse,
   nthreads,
   height, width,
   strideY, strideX);

  hipError_t status = hipPeekAtLastError() ;
  return (status == hipSuccess) ? vl::vlSuccess : vl::vlErrorCuda ;
}

